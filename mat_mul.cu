#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include "hip/hip_runtime.h"
#include ""

#define TILE_SIZE 32  // Tile size for shared memory
#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(-1); \
    } \
} while (0)

struct Kernel {
    float* A, * B, * C;
    std::string name;
    void (*kernel)(float*, float*, float*, int);
    int N;
    float expected_value;
};

struct KernelInfo {
    std::string name;
    void (*kernel)(float*, float*, float*, int);
    int N;
    float expected_value;
};


__global__
void matrixAdd(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        C[row * N + col] = A[row * N + col] + B[row * N + col];
    }
}

__global__
void matrixMultiply(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; k++) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

__global__
void matrixMultiplyOptimized(float* A, float* B, float* C, int N) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float value = 0.0f;

    for (int i = 0; i < (N + TILE_SIZE - 1) / TILE_SIZE; i++) {
        if (row < N && i * TILE_SIZE + threadIdx.x < N)
            tileA[threadIdx.y][threadIdx.x] = A[row * N + i * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && i * TILE_SIZE + threadIdx.y < N)
            tileB[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * N + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++) {
            value += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

void allocateAndInitializeMatrix(float** A, float** B, float** C, int N) {
    size_t size = N * N * sizeof(float);
    CUDA_CHECK(hipMallocManaged(A, size));
    CUDA_CHECK(hipMallocManaged(B, size));
    CUDA_CHECK(hipMallocManaged(C, size));

    for (int i = 0; i < N * N; i++) {
        (*A)[i] = 1.0f;
        (*B)[i] = 2.0f;
    }
}

float calculateMaxError(float* C, int N, float expected_value) {
    float maxError = 0.0f;
    for (int i = 0; i < N * N; i++) {
        maxError = fmax(maxError, fabs(C[i] - expected_value));
    }
    return maxError;
}

void executeKernel(const Kernel& data) {
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((data.N + blockSize.x - 1) / blockSize.x, (data.N + blockSize.y - 1) / blockSize.y);
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Record start time
    CUDA_CHECK(hipEventRecord(start, stream));

    // Launch kernel
    data.kernel << <gridSize, blockSize, 0, stream >> > (data.A, data.B, data.C, data.N);
    CUDA_CHECK(hipGetLastError());

    // Record stop time
    CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));

    // Compute elapsed time
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    // Destroy events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipStreamDestroy(stream));

    // Output time for each kernel
    std::cout << data.name << " | Execution Time: " << milliseconds << " ms | Max Error: "
        << calculateMaxError(data.C, data.N, data.expected_value) << std::endl;
}

__global__ void warmupKernel() {}

int main(void) {
    hipFree(0);
    warmupKernel << <1, 1 >> > ();
    hipDeviceSynchronize();

    std::vector<KernelInfo> kernel_config_vec = {
        {"Matrix Multiplication Optimized #1", matrixMultiplyOptimized, 1 << 11, ((int) 1 << 11) * 2.0f},
        {"Matrix Multiplication Optimized #2", matrixMultiplyOptimized, 1 << 11, ((int) 1 << 11) * 2.0f},
        {"Matrix Multiplication Optimized #3", matrixMultiplyOptimized, 1 << 11, ((int) 1 << 11) * 2.0f},
        {"Matrix Multiplication #1", matrixMultiply, 1 << 11, ((int) 1 << 11) * 2.0f},
        {"Matrix Add #1", matrixAdd, 1 << 11, 3.0f},

    };

    for (const auto& kernel : kernel_config_vec) {
        float* A, * B, * C;
        allocateAndInitializeMatrix(&A, &B, &C, kernel.N);
        executeKernel({ A, B, C, kernel.name, kernel.kernel, kernel.N, kernel.expected_value});

        hipFree(A);
        hipFree(B);
        hipFree(C);
    }

    return 0;
}